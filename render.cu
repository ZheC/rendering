#include "hip/hip_runtime.h"
// -*- mode: c++ -*-

// Produce renders of CAD model

// include DevIL for image loading
#include <IL/il.h>
// auxiliary C file to read the shader text files
#include "textfile.h"
// assimp
#include "assimp/Importer.hpp"	
#include "assimp/postprocess.h"
#include "assimp/scene.h"
//opencv
#include "cv.h"
#include "highgui.h"

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <math.h>
#include <fstream>
#include <map>
#include <string>
#include <vector>

char gvar_vertex_fname[256];
char gvar_fragment_fname[256];
char gvar_model_fname[256];

float gvar_delta_rot_x, gvar_delta_rot_y, gvar_delta_rot_z;
int gvar_num_rot_x, gvar_num_rot_y, gvar_num_rot_z;

float gvar_proj_mtx_horiz_fov;
float gvar_proj_mtx_near_clip_plane, gvar_proj_mtx_far_clip_plane;
int gvar_render_size_width, gvar_render_size_height;

/// params for model -- default values below, can be overridden
#define MODEL_FILENAME "./input/17_Pet_bottle_pet_tea.ply"
//#define MODEL_FILENAME "./input/19_Dish_bawl_rice.ply"
//#define MODEL_FILENAME "./input/14_MugCup_green.ply"
//#define MODEL_FILENAME "./input/arc.obj" // this can be OBJ or PLY (may be others work as well)

#define VERTEX_FILENAME "./input/with_texture.vert" // these have to do w/ shading, etc. (not the actual object); more details here: http://stackoverflow.com/questions/6432838/what-is-the-correct-file-extension-for-glsl-shaders
#define FRAGMENT_FILENAME "./input/with_texture.frag"

/// params for render angles -- default values below, can be overridden
#define DELTA_ROT_X 20 // deg.
#define DELTA_ROT_Y 20 // deg.
#define DELTA_ROT_Z 10 // deg.
#define NUM_ROT_X 2 // in increments of DELTA_ROT_X
#define NUM_ROT_Y 3 // in increments of DELTA_ROT_Y
#define NUM_ROT_Z 36 // in increments of DELTA_ROT_Z

/// params for projection matrix, image size, etc. in renders (more details on the whole process of rendering here: http://www.opengl-tutorial.org/beginners-tutorials/tutorial-3-matrices/)
// FOV and RENDER_SIZE basically make up the camera focal length (radial distortion probably not bothered with)
#define PROJ_MTX_HORIZONTAL_FOV 20.0f // as the FOV increases (or decreases), the object is further from (or closer to) the camera
#define PROJ_MTX_NEAR_CLIP_PLANE 0.1f
#define PROJ_MTX_FAR_CLIP_PLANE 100.0f
#define RENDER_SIZE_WIDTH 180
#define RENDER_SIZE_HEIGHT RENDER_SIZE_WIDTH 

aiVector3D scene_min, scene_max, scene_center;

/// Information to render each assimp node
struct MyMesh{

  GLuint vao;
  GLuint texIndex;
  GLuint uniformBlockIndex;
  int numFaces;
};

///
std::vector<struct MyMesh> myMeshes;

/// This is for a shader uniform block
struct MyMaterial{

  float diffuse[4];
  float ambient[4];
  float specular[4];
  float emissive[4];
  float shininess;
  int texCount;
};

/// Model Matrix (part of the OpenGL Model View Matrix)
float modelMatrix[16];

/// For push and pop matrix
std::vector<float *> matrixStack;

/// Vertex Attribute Locations
GLuint vertexLoc=0, normalLoc=1, texCoordLoc=2;

/// Uniform Bindings Points
GLuint matricesUniLoc = 1, materialUniLoc = 2;

/// The sampler uniform for textured models
// we are assuming a single texture so this will
//always be texture unit 0
GLuint texUnit = 0;

/// Uniform Buffer for Matrices
// this buffer will contain 3 matrices: projection, view and model
// each matrix is a float array with 16 components
GLuint matricesUniBuffer;
#define MatricesUniBufferSize sizeof(float) * 16 * 3
#define ProjMatrixOffset 0
#define ViewMatrixOffset sizeof(float) * 16
#define ModelMatrixOffset sizeof(float) * 16 * 2
#define MatrixSize sizeof(float) * 16

/// Program and Shader Identifiers
GLuint program, vertexShader, fragmentShader;

// Shader Names
//char *vertexFileName = "dirlightdiffambpix.vert";
//char *fragmentFileName = "dirlightdiffambpix.frag";
//char *vertexFileName = "with_texture.vert";
//char *fragmentFileName = "with_texture.frag";

/// Create an instance of the Importer class
Assimp::Importer importer;

/// the global Assimp scene object
const aiScene* scene = NULL;

/// scale factor for the model to fit in the window
float scaleFactor;

/// images / texture
// map image filenames to textureIds pointer to texture Array
std::map<std::string, GLuint> textureIdMap;	

// Replace the model name by your model's filename
//static const std::string modelname = "arc.obj";

/// Camera Position
/// @warning why is camZ fixed?
float camX = 0, camY = 0, camZ = 5;

/// Mouse Tracking Variables
int startX, startY, tracking = 0;

/// Camera Spherical Coordinates
float alpha = 0.0f, beta = 0.0f;
float r = 5.0f;

///
static inline float 
DegToRad(float degrees) 
{ 
  return (float)(degrees * (M_PI / 180.0f));
};

// ----------------------------------------------------
// VECTOR STUFF

/// res = a cross b;
void crossProduct( float *a, float *b, float *res) {
  res[0] = a[1] * b[2]  -  b[1] * a[2];
  res[1] = a[2] * b[0]  -  b[2] * a[0];
  res[2] = a[0] * b[1]  -  b[0] * a[1];
}

/// Normalize a vec3
void normalize(float *a) {
  float mag = sqrt(a[0] * a[0]  +  a[1] * a[1]  +  a[2] * a[2]);
  a[0] /= mag;
  a[1] /= mag;
  a[2] /= mag;
}

// ----------------------------------------------------
// MATRIX STUFF

/// Push for modelMatrix
void pushMatrix() {

  float *aux = (float *)malloc(sizeof(float) * 16);
  memcpy(aux, modelMatrix, sizeof(float) * 16);
  matrixStack.push_back(aux);
}

/// Pop for modelMatrix
void popMatrix() {

  float *m = matrixStack[matrixStack.size()-1];
  memcpy(modelMatrix, m, sizeof(float) * 16);
  matrixStack.pop_back();
  free(m);
}

/// sets the square matrix mat to the identity matrix,
// size refers to the number of rows (or columns)
void setIdentityMatrix( float *mat, int size) {

  // fill matrix with 0s
  for (int i = 0; i < size * size; ++i)
    mat[i] = 0.0f;

  // fill diagonal with 1s
  for (int i = 0; i < size; ++i)
    mat[i + i * size] = 1.0f;
}

/// a = a * b;
void multMatrix(float *a, float *b) {

  float res[16];

  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 4; ++j) {
      res[j*4 + i] = 0.0f;
      for (int k = 0; k < 4; ++k) {
	res[j*4 + i] += a[k*4 + i] * b[j*4 + k]; 
      }
    }
  }
  memcpy(a, res, 16 * sizeof(float));
}

/// Defines a transformation matrix mat with a translation
void setTranslationMatrix(float *mat, float x, float y, float z) {

  setIdentityMatrix(mat,4);
  mat[12] = x;
  mat[13] = y;
  mat[14] = z;
}

/// Defines a transformation matrix mat with a scale
void setScaleMatrix(float *mat, float sx, float sy, float sz) {

  setIdentityMatrix(mat,4);
  mat[0] = sx;
  mat[5] = sy;
  mat[10] = sz;
}

/// Defines a transformation matrix mat with a rotation 
// angle alpha and a rotation axis (x,y,z)
void setRotationMatrix(float *mat, float angle, float x, float y, float z) {

  float radAngle = DegToRad(angle);
  float co = cos(radAngle);
  float si = sin(radAngle);
  float x2 = x*x;
  float y2 = y*y;
  float z2 = z*z;

  mat[0] = x2 + (y2 + z2) * co; 
  mat[4] = x * y * (1 - co) - z * si;
  mat[8] = x * z * (1 - co) + y * si;
  mat[12]= 0.0f;
	   
  mat[1] = x * y * (1 - co) + z * si;
  mat[5] = y2 + (x2 + z2) * co;
  mat[9] = y * z * (1 - co) - x * si;
  mat[13]= 0.0f;
	   
  mat[2] = x * z * (1 - co) - y * si;
  mat[6] = y * z * (1 - co) + x * si;
  mat[10]= z2 + (x2 + y2) * co;
  mat[14]= 0.0f;
	   
  mat[3] = 0.0f;
  mat[7] = 0.0f;
  mat[11]= 0.0f;
  mat[15]= 1.0f;
}

// ----------------------------------------------------
/// Model Matrix 

/// Copies the modelMatrix to the uniform buffer
void setModelMatrix() {
  glBindBuffer(GL_UNIFORM_BUFFER,matricesUniBuffer);
  glBufferSubData(GL_UNIFORM_BUFFER,ModelMatrixOffset, MatrixSize, modelMatrix);
  glBindBuffer(GL_UNIFORM_BUFFER,0);
}

/// The equivalent to glTranslate applied to the model matrix
void translate(float x, float y, float z) {
  float aux[16];
  setTranslationMatrix(aux,x,y,z);
  multMatrix(modelMatrix,aux);
  setModelMatrix();
}

/// The equivalent to glRotate applied to the model matrix
void rotate(float angle, float x, float y, float z) {
  float aux[16];
  setRotationMatrix(aux,angle,x,y,z);
  multMatrix(modelMatrix,aux);
  setModelMatrix();
}

/// The equivalent to glScale applied to the model matrix
void scale(float x, float y, float z) {
  float aux[16];
  setScaleMatrix(aux,x,y,z);
  multMatrix(modelMatrix,aux);
  setModelMatrix();
}

// ----------------------------------------------------
/// Projection Matrix 

/// Computes the projection Matrix and stores it in the uniform buffer
void buildProjectionMatrix(float fov, float ratio, float nearp, float farp) {
  float projMatrix[16];
  float f = 1.0f / tan (fov * (M_PI / 360.0f));
  setIdentityMatrix(projMatrix,4);

  projMatrix[0] = f / ratio;
  projMatrix[1 * 4 + 1] = f;
  projMatrix[2 * 4 + 2] = (farp + nearp) / (nearp - farp);
  projMatrix[3 * 4 + 2] = (2.0f * farp * nearp) / (nearp - farp);
  projMatrix[2 * 4 + 3] = -1.0f;
  projMatrix[3 * 4 + 3] = 0.0f;

  glBindBuffer(GL_UNIFORM_BUFFER,matricesUniBuffer);
  glBufferSubData(GL_UNIFORM_BUFFER, ProjMatrixOffset, MatrixSize, projMatrix);
  glBindBuffer(GL_UNIFORM_BUFFER,0);

}

// ----------------------------------------------------
/// View Matrix

/// Computes the viewMatrix and stores it in the uniform buffer
void setCamera(float posX, float posY, float posZ, 
	       float lookAtX, float lookAtY, float lookAtZ) {

  float dir[3], right[3], up[3];
  up[0] = 0.0f;	up[1] = 1.0f;	up[2] = 0.0f;

  dir[0] =  (lookAtX - posX);
  dir[1] =  (lookAtY - posY);
  dir[2] =  (lookAtZ - posZ);
  normalize(dir);

  crossProduct(dir,up,right);
  normalize(right);

  crossProduct(right,dir,up);
  normalize(up);

  float viewMatrix[16],aux[16];

  viewMatrix[0]  = right[0];
  viewMatrix[4]  = right[1];
  viewMatrix[8]  = right[2];
  viewMatrix[12] = 0.0f;

  viewMatrix[1]  = up[0];
  viewMatrix[5]  = up[1];
  viewMatrix[9]  = up[2];
  viewMatrix[13] = 0.0f;

  viewMatrix[2]  = -dir[0];
  viewMatrix[6]  = -dir[1];
  viewMatrix[10] = -dir[2];
  viewMatrix[14] =  0.0f;

  viewMatrix[3]  = 0.0f;
  viewMatrix[7]  = 0.0f;
  viewMatrix[11] = 0.0f;
  viewMatrix[15] = 1.0f;

  setTranslationMatrix(aux, -posX, -posY, -posZ);

  multMatrix(viewMatrix, aux);
	
  glBindBuffer(GL_UNIFORM_BUFFER, matricesUniBuffer);
  glBufferSubData(GL_UNIFORM_BUFFER, ViewMatrixOffset, MatrixSize, viewMatrix);
  glBindBuffer(GL_UNIFORM_BUFFER,0);
}

// ----------------------------------------------------------------------------

#define aisgl_min(x,y) (x<y?x:y)
#define aisgl_max(x,y) (y>x?y:x)

///
void get_bounding_box_for_node (const aiNode* nd, 
				aiVector3D* min, 
				aiVector3D* max)
{
  aiMatrix4x4 prev;
  unsigned int n = 0, t;

  for (; n < nd->mNumMeshes; ++n) {
    const aiMesh* mesh = scene->mMeshes[nd->mMeshes[n]];
    for (t = 0; t < mesh->mNumVertices; ++t) {

      aiVector3D tmp = mesh->mVertices[t];

      min->x = aisgl_min(min->x,tmp.x);
      min->y = aisgl_min(min->y,tmp.y);
      min->z = aisgl_min(min->z,tmp.z);

      max->x = aisgl_max(max->x,tmp.x);
      max->y = aisgl_max(max->y,tmp.y);
      max->z = aisgl_max(max->z,tmp.z);
    }
  }

  for (n = 0; n < nd->mNumChildren; ++n) {
    get_bounding_box_for_node(nd->mChildren[n],min,max);
  }
}

///
void get_bounding_box (aiVector3D* min, aiVector3D* max)
{

  min->x = min->y = min->z =  1e10f;
  max->x = max->y = max->z = -1e10f;
  get_bounding_box_for_node(scene->mRootNode,min,max);
}

///
bool Import3DFromFile( const std::string& pFile)
{
  //check if file exists
  std::ifstream fin(pFile.c_str());
  if(!fin.fail()) {
    fin.close();
  }
  else{
    printf("Couldn't open file: %s\n", pFile.c_str());
    printf("%s\n", importer.GetErrorString());
    return false;
  }

  scene = importer.ReadFile( pFile, aiProcessPreset_TargetRealtime_Quality);

  // If the import failed, report it
  if( !scene)
    {
      printf("%s\n", importer.GetErrorString());
      return false;
    }

  printf("Import of scene %s succeeded\n",pFile.c_str());

  get_bounding_box(&scene_min, &scene_max);
  scene_center.x = (scene_min.x + scene_max.x) / 2.0f;
  scene_center.y = (scene_min.y + scene_max.y) / 2.0f;
  scene_center.z = (scene_min.z + scene_max.z) / 2.0f;
  printf("Scene center: %f, %f, %f\n", scene_center.x, scene_center.y, scene_center.z);
  //center the model
  glTranslatef( -scene_center.x, -scene_center.y, -scene_center.z );
  //translate( -scene_center.x, -scene_center.y, -1500 );
		
  float tmp;
  tmp = scene_max.x-scene_min.x;
  tmp = scene_max.y - scene_min.y > tmp?scene_max.y - scene_min.y:tmp;
  tmp = scene_max.z - scene_min.z > tmp?scene_max.z - scene_min.z:tmp;
  scaleFactor = 1.4f / tmp;

  return true;
}

///
int LoadGLTextures(const aiScene* scene)
{
  ILboolean success;

  /* initialization of DevIL */
  ilInit(); 

  /* scan scene's materials for textures */
  for (unsigned int m=0; m<scene->mNumMaterials; ++m)
    {
      int texIndex = 0;
      aiString path;	// filename

      aiReturn texFound = scene->mMaterials[m]->GetTexture(aiTextureType_DIFFUSE, texIndex, &path);
      while (texFound == AI_SUCCESS) {
	//fill map with textures, OpenGL image ids set to 0
	textureIdMap[path.data] = 0; 
	// more textures?
	texIndex++;
	texFound = scene->mMaterials[m]->GetTexture(aiTextureType_DIFFUSE, texIndex, &path);
      }
    }

  int numTextures = textureIdMap.size();

  /* create and fill array with DevIL texture ids */
  ILuint* imageIds = new ILuint[numTextures];
  ilGenImages(numTextures, imageIds); 

  /* create and fill array with GL texture ids */
  GLuint* textureIds = new GLuint[numTextures];
  glGenTextures(numTextures, textureIds); /* Texture name generation */

  /* get iterator */
  std::map<std::string, GLuint>::iterator itr = textureIdMap.begin();
  int i=0;
  for (; itr != textureIdMap.end(); ++i, ++itr)
    {
      //save IL image ID
      std::string filename = (*itr).first;  // get filename
      (*itr).second = textureIds[i];	  // save texture id for filename in map

      ilBindImage(imageIds[i]); /* Binding of DevIL image name */
      ilEnable(IL_ORIGIN_SET);
      ilOriginFunc(IL_ORIGIN_LOWER_LEFT); 
      success = ilLoadImage((ILstring)filename.c_str());

      if (success) {
	/* Convert image to RGBA */
	ilConvertImage(IL_RGBA, IL_UNSIGNED_BYTE); 

	/* Create and load textures to OpenGL */
	glBindTexture(GL_TEXTURE_2D, textureIds[i]); 
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR); 
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, ilGetInteger(IL_IMAGE_WIDTH),
		     ilGetInteger(IL_IMAGE_HEIGHT), 0, GL_RGBA, GL_UNSIGNED_BYTE,
		     ilGetData()); 
      }
      else 
	printf("Couldn't load Image: %s\n", filename.c_str());
    }

  ilDeleteImages(numTextures, imageIds); 

  //Cleanup
  delete [] imageIds;
  delete [] textureIds;

  //return success;
  return true;
}

///
void set_float4(float f[4], float a, float b, float c, float d)
{
  f[0] = a;
  f[1] = b;
  f[2] = c;
  f[3] = d;
}

///
void color4_to_float4(const aiColor4D *c, float f[4])
{
  f[0] = c->r;
  f[1] = c->g;
  f[2] = c->b;
  f[3] = c->a;
}

///
void genVAOsAndUniformBuffer(const aiScene *sc) {

  struct MyMesh aMesh;
  struct MyMaterial aMat; 
  GLuint buffer;
	
  // For each mesh
  for (unsigned int n = 0; n < sc->mNumMeshes; ++n)
    {
      const aiMesh* mesh = sc->mMeshes[n];

      // create array with faces
      // have to convert from Assimp format to array
      unsigned int *faceArray;
      faceArray = (unsigned int *)malloc(sizeof(unsigned int) * mesh->mNumFaces * 3);
      unsigned int faceIndex = 0;

      for (unsigned int t = 0; t < mesh->mNumFaces; ++t) {
	const aiFace* face = &mesh->mFaces[t];

	memcpy(&faceArray[faceIndex], face->mIndices,3 * sizeof(unsigned int));
	faceIndex += 3;
      }
      aMesh.numFaces = sc->mMeshes[n]->mNumFaces;

      // generate Vertex Array for mesh
      glGenVertexArrays(1,&(aMesh.vao));
      glBindVertexArray(aMesh.vao);

      // buffer for faces
      glGenBuffers(1, &buffer);
      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, buffer);
      glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(unsigned int) * mesh->mNumFaces * 3, faceArray, GL_STATIC_DRAW);

      // buffer for vertex positions
      if (mesh->HasPositions()) {
	glGenBuffers(1, &buffer);
	glBindBuffer(GL_ARRAY_BUFFER, buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float)*3*mesh->mNumVertices, mesh->mVertices, GL_STATIC_DRAW);
	glEnableVertexAttribArray(vertexLoc);
	glVertexAttribPointer(vertexLoc, 3, GL_FLOAT, 0, 0, 0);
      }

      // buffer for vertex normals
      if (mesh->HasNormals()) {
	glGenBuffers(1, &buffer);
	glBindBuffer(GL_ARRAY_BUFFER, buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float)*3*mesh->mNumVertices, mesh->mNormals, GL_STATIC_DRAW);
	glEnableVertexAttribArray(normalLoc);
	glVertexAttribPointer(normalLoc, 3, GL_FLOAT, 0, 0, 0);
      }

      // buffer for vertex texture coordinates
      if (mesh->HasTextureCoords(0)) {
	float *texCoords = (float *)malloc(sizeof(float)*2*mesh->mNumVertices);
	for (unsigned int k = 0; k < mesh->mNumVertices; ++k) {

	  texCoords[k*2]   = mesh->mTextureCoords[0][k].x;
	  texCoords[k*2+1] = mesh->mTextureCoords[0][k].y; 
				
	}
	glGenBuffers(1, &buffer);
	glBindBuffer(GL_ARRAY_BUFFER, buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float)*2*mesh->mNumVertices, texCoords, GL_STATIC_DRAW);
	glEnableVertexAttribArray(texCoordLoc);
	glVertexAttribPointer(texCoordLoc, 2, GL_FLOAT, 0, 0, 0);
      }

      // unbind buffers
      glBindVertexArray(0);
      glBindBuffer(GL_ARRAY_BUFFER,0);
      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER,0);
	
      // create material uniform buffer
      aiMaterial *mtl = sc->mMaterials[mesh->mMaterialIndex];
			
      aiString texPath;	//contains filename of texture
      if(AI_SUCCESS == mtl->GetTexture(aiTextureType_DIFFUSE, 0, &texPath)){
	//bind texture
	unsigned int texId = textureIdMap[texPath.data];
	aMesh.texIndex = texId;
	aMat.texCount = 1;
      }
      else
	aMat.texCount = 0;

      float c[4];
      set_float4(c, 0.8f, 0.8f, 0.8f, 1.0f);
      aiColor4D diffuse;
      if(AI_SUCCESS == aiGetMaterialColor(mtl, AI_MATKEY_COLOR_DIFFUSE, &diffuse))
	color4_to_float4(&diffuse, c);
      memcpy(aMat.diffuse, c, sizeof(c));

      set_float4(c, 0.2f, 0.2f, 0.2f, 1.0f);
      aiColor4D ambient;
      if(AI_SUCCESS == aiGetMaterialColor(mtl, AI_MATKEY_COLOR_AMBIENT, &ambient))
	color4_to_float4(&ambient, c);
      memcpy(aMat.ambient, c, sizeof(c));

      set_float4(c, 0.0f, 0.0f, 0.0f, 1.0f);
      aiColor4D specular;
      if(AI_SUCCESS == aiGetMaterialColor(mtl, AI_MATKEY_COLOR_SPECULAR, &specular))
	color4_to_float4(&specular, c);
      memcpy(aMat.specular, c, sizeof(c));

      set_float4(c, 0.0f, 0.0f, 0.0f, 1.0f);
      aiColor4D emission;
      if(AI_SUCCESS == aiGetMaterialColor(mtl, AI_MATKEY_COLOR_EMISSIVE, &emission))
	color4_to_float4(&emission, c);
      memcpy(aMat.emissive, c, sizeof(c));

      float shininess = 0.0;
      unsigned int max;
      aiGetMaterialFloatArray(mtl, AI_MATKEY_SHININESS, &shininess, &max);
      aMat.shininess = shininess;

      glGenBuffers(1,&(aMesh.uniformBlockIndex));
      glBindBuffer(GL_UNIFORM_BUFFER,aMesh.uniformBlockIndex);
      glBufferData(GL_UNIFORM_BUFFER, sizeof(aMat), (void *)(&aMat), GL_STATIC_DRAW);

      myMeshes.push_back(aMesh);
    }
}

// ------------------------------------------------------------
/// Reshape Callback Function
void changeSize(int w, int h) {

  float ratio;
  // Prevent a divide by zero, when window is too short
  // (you cant make a window of zero width).
  if(h == 0)
    h = 1;

  // Set the viewport to be the entire window
  glViewport(0, 0, w, h);

  ratio = (1.0f * w) / h;
  buildProjectionMatrix(gvar_proj_mtx_horiz_fov,\
			ratio,\
			gvar_proj_mtx_near_clip_plane,\
			gvar_proj_mtx_far_clip_plane);
}

// ------------------------------------------------------------
/// Render stuff

/// Render Assimp Model
void recursive_render (const aiScene *sc, const aiNode* nd)
{
  // Get node transformation matrix
  aiMatrix4x4 m = nd->mTransformation;
  // OpenGL matrices are column major
  m.Transpose();

  // save model matrix and apply node transformation
  pushMatrix();

  float aux[16];
  memcpy(aux,&m,sizeof(float) * 16);
  multMatrix(modelMatrix, aux);
  setModelMatrix();

  // draw all meshes assigned to this node
  for (unsigned int n=0; n < nd->mNumMeshes; ++n){
    // bind material uniform
    glBindBufferRange(GL_UNIFORM_BUFFER, materialUniLoc, myMeshes[nd->mMeshes[n]].uniformBlockIndex, 0, sizeof(struct MyMaterial));	
    // bind texture
    glBindTexture(GL_TEXTURE_2D, myMeshes[nd->mMeshes[n]].texIndex);
    // bind VAO
    glBindVertexArray(myMeshes[nd->mMeshes[n]].vao);
    // draw
    glDrawElements(GL_TRIANGLES,myMeshes[nd->mMeshes[n]].numFaces*3,GL_UNSIGNED_INT,0);

  }

  // draw all children
  for (unsigned int n=0; n < nd->mNumChildren; ++n){
    recursive_render(sc, nd->mChildren[n]);
  }
  popMatrix();
}

/// Rendering Callback Function
void renderScene(void) {

  int i,j,k;
  for(i=-1;i<gvar_num_rot_x;i++){ // @todo, why does this start at -1 but others start at 0?  affects exit condition of loop
    for(j=0;j<gvar_num_rot_y;j++){
      for(k=0;k<gvar_num_rot_z;k++){
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set camera matrix
	setCamera(camX,camY,camZ,0,0,0);

	// set the model matrix to the identity Matrix
	setIdentityMatrix(modelMatrix,4);
	
	// sets the model matrix to a scale matrix so that the model fits in the window; scaleFactor is determined by looking at the size of the object and trying to fit the object in the window
	scale(scaleFactor, scaleFactor, scaleFactor);
	
    	//rotate(90,0.f,0.f,1.f);
    	rotate(-90,1.f,0.f,0.f);
    	rotate(-90,0.f,0.f,1.f);
    
	rotate(gvar_delta_rot_x*i,1.f,0.f,0.f); // rotate it around the x axis	
	rotate(gvar_delta_rot_y*j,0.f,1.f,0.f); // rotate it around the y axis
	rotate(gvar_delta_rot_z*k,0.f,0.f,1.f); // rotate it around the z axis

	// use our shader
	glUseProgram(program);
	glUniform1i(texUnit,0);

	recursive_render(scene, scene->mRootNode);

	// swap buffers
	glutSwapBuffers();

	int w=gvar_render_size_width,h=gvar_render_size_height;	//save color image    
	char filename[50];
	/*IplImage* src=cvCreateImage(cvSize(w,h), IPL_DEPTH_8U,1);   //save depth image
	  glPixelStorei(GL_PACK_ALIGNMENT, 1);
	  glPixelStorei(GL_PACK_ROW_LENGTH, 0);
	  glReadPixels(0, 0, w, h, GL_DEPTH_COMPONENT, GL_UNSIGNED_BYTE, src->imageData);
	  cvFlip(src, src, 0);      //flip image in x axes	
	  sprintf(filename, "d_%02d_%02d.png",i,k);
	  cvSaveImage(filename,src,0);*/
	
	IplImage* img=cvCreateImage(cvSize(w,h), IPL_DEPTH_8U,3);
    	glPixelStorei(GL_PACK_ALIGNMENT, 1);
    	glPixelStorei(GL_PACK_ROW_LENGTH, 0);
    	glReadPixels(0, 0, w, h, GL_BGR_EXT, GL_UNSIGNED_BYTE, img->imageData);
	cvFlip(img, img, 0);	
    	sprintf(filename, "./output/c_%02d_%02d_%02d.png",i+1,j,k);
	cvSaveImage(filename,img);
		
      }
    }

    //printf( "i %d j %d k %d\n", i, j, k );
    
    if ( i == gvar_num_rot_x-1 &&
	 j == gvar_num_rot_y &&
	 k == gvar_num_rot_z ) {
      printf( "Exiting loop\n" );
      throw 999; // Trick to exit glutMainLoop from https://www.opengl.org/discussion_boards/showthread.php/166643-how-to-come-out-from-glutMainLoop
      //exit(0);
    }

  }

}

// --------------------------------------------------------
/// Shader Stuff

///
void printShaderInfoLog(GLuint obj)
{
  int infologLength = 0;
  int charsWritten  = 0;
  char *infoLog;

  glGetShaderiv(obj, GL_INFO_LOG_LENGTH,&infologLength);

  if (infologLength > 0)
    {
      infoLog = (char *)malloc(infologLength);
      glGetShaderInfoLog(obj, infologLength, &charsWritten, infoLog);
      printf("%s\n",infoLog);
      free(infoLog);
    }
}

///
void printProgramInfoLog(GLuint obj)
{
  int infologLength = 0;
  int charsWritten  = 0;
  char *infoLog;

  glGetProgramiv(obj, GL_INFO_LOG_LENGTH,&infologLength);

  if (infologLength > 0)
    {
      infoLog = (char *)malloc(infologLength);
      glGetProgramInfoLog(obj, infologLength, &charsWritten, infoLog);
      printf("%s\n",infoLog);
      free(infoLog);
    }
}

///
GLuint setupShaders() {
  char *vs = NULL,*fs = NULL;
  GLuint p,v,f;

  v = glCreateShader(GL_VERTEX_SHADER);
  f = glCreateShader(GL_FRAGMENT_SHADER);

  vs = textFileRead( gvar_vertex_fname /*VERTEX_FILENAME*/ );
  fs = textFileRead( gvar_fragment_fname /*FRAGMENT_FILENAME*/ );

  const char * vv = vs;
  const char * ff = fs;

  glShaderSource(v, 1, &vv,NULL);
  glShaderSource(f, 1, &ff,NULL);

  free(vs);free(fs);

  glCompileShader(v);
  glCompileShader(f);

  printShaderInfoLog(v);
  printShaderInfoLog(f);

  p = glCreateProgram();
  glAttachShader(p,v);
  glAttachShader(p,f);

  glBindFragDataLocation(p, 0, "output");

  glBindAttribLocation(p,vertexLoc,"position");
  glBindAttribLocation(p,normalLoc,"normal");
  glBindAttribLocation(p,texCoordLoc,"texCoord");

  glLinkProgram(p);
  glValidateProgram(p);
  printProgramInfoLog(p);

  program = p;
  vertexShader = v;
  fragmentShader = f;
	
  GLuint k = glGetUniformBlockIndex(p,"Matrices");
  glUniformBlockBinding(p, k, matricesUniLoc);
  glUniformBlockBinding(p, glGetUniformBlockIndex(p,"Material"), materialUniLoc);

  texUnit = glGetUniformLocation(p,"texUnit");

  return(p);
}

// ------------------------------------------------------------
/// Model loading and OpenGL setup

///
int init()					 
{
  if (!Import3DFromFile( gvar_model_fname /*MODEL_FILENAME*/ )) 
    return(0);

  LoadGLTextures(scene);

  glGetUniformBlockIndex = (PFNGLGETUNIFORMBLOCKINDEXPROC) glutGetProcAddress("glGetUniformBlockIndex");
  glUniformBlockBinding = (PFNGLUNIFORMBLOCKBINDINGPROC) glutGetProcAddress("glUniformBlockBinding");
  glGenVertexArrays = (PFNGLGENVERTEXARRAYSPROC) glutGetProcAddress("glGenVertexArrays");
  glBindVertexArray = (PFNGLBINDVERTEXARRAYPROC)glutGetProcAddress("glBindVertexArray");
  glBindBufferRange = (PFNGLBINDBUFFERRANGEPROC) glutGetProcAddress("glBindBufferRange");
  glDeleteVertexArrays = (PFNGLDELETEVERTEXARRAYSPROC) glutGetProcAddress("glDeleteVertexArrays");

  program = setupShaders();
  genVAOsAndUniformBuffer(scene);

  glEnable(GL_DEPTH_TEST);		
  glClearColor(0.0f, 0.0f, 0.0f, 0.0f);

  glGenBuffers(1,&matricesUniBuffer);
  glBindBuffer(GL_UNIFORM_BUFFER, matricesUniBuffer);
  glBufferData(GL_UNIFORM_BUFFER, MatricesUniBufferSize,NULL,GL_DYNAMIC_DRAW);
  glBindBufferRange(GL_UNIFORM_BUFFER, matricesUniLoc, matricesUniBuffer, 0, MatricesUniBufferSize);	//setUniforms();
  glBindBuffer(GL_UNIFORM_BUFFER,0);

  glEnable(GL_MULTISAMPLE);
  return true;					
}

// ------------------------------------------------------------
/// Main function
int main(int argc, char **argv) {
  // set default param. values
  strcpy( gvar_vertex_fname, VERTEX_FILENAME );
  strcpy( gvar_fragment_fname, FRAGMENT_FILENAME );
  strcpy( gvar_model_fname, MODEL_FILENAME );

  gvar_delta_rot_x = DELTA_ROT_X;
  gvar_delta_rot_y = DELTA_ROT_Y;
  gvar_delta_rot_z = DELTA_ROT_Z;
  gvar_num_rot_x = NUM_ROT_X;
  gvar_num_rot_y = NUM_ROT_Y;
  gvar_num_rot_z = NUM_ROT_Z;

  gvar_proj_mtx_horiz_fov = PROJ_MTX_HORIZONTAL_FOV;
  gvar_proj_mtx_near_clip_plane = PROJ_MTX_NEAR_CLIP_PLANE;
  gvar_proj_mtx_far_clip_plane = PROJ_MTX_FAR_CLIP_PLANE;
  gvar_render_size_width = RENDER_SIZE_WIDTH;
  gvar_render_size_height = RENDER_SIZE_HEIGHT;

  // @todo take param's from cmd line or config file

  printf( "vertex, fragment, and model filenames: %s, %s, %s\n",\
	  gvar_vertex_fname, gvar_fragment_fname, gvar_model_fname );
  printf ( "Delta rotation x, y, z (deg.): %.2g, %.2g, %.2g\n",
	   gvar_delta_rot_x, gvar_delta_rot_y, gvar_delta_rot_z );
  printf( "Number of rotations: %d, %d, %d\n",\
	  gvar_num_rot_x, gvar_num_rot_y, gvar_num_rot_z );
  // @todo figure out what angle ranges are covered with these param's (need to know start/stop conditions of code)
  printf( "Projection matrix horizontal FOV, near clip plane, and far clip plane; render size width, render size height: %.2g, %.2g, %.2g, %d, %d\n",\
	  gvar_proj_mtx_horiz_fov, gvar_proj_mtx_near_clip_plane, gvar_proj_mtx_far_clip_plane, gvar_render_size_width, gvar_render_size_height);

  //printf ("--> Close graphics window to quit program <--\n\n" );

  try {

  glutInit(&argc, argv);

  glutInitDisplayMode(GLUT_DEPTH|GLUT_DOUBLE|GLUT_RGBA|GLUT_MULTISAMPLE);

  glutInitContextVersion (3, 3);
  glutInitContextFlags (GLUT_COMPATIBILITY_PROFILE );

  glutInitWindowPosition(100,100);
  glutInitWindowSize(gvar_render_size_width,gvar_render_size_height);
  glutCreateWindow("Model");
		
  //Callback Registration
  glutDisplayFunc(renderScene);
  glutReshapeFunc(changeSize);
  //glutIdleFunc(renderScene);

  //Init GLEW
  glewInit();
  if (!glewIsSupported("GL_VERSION_3_3")){
    printf("OpenGL 3.3 not supported\n");
    return(1);
  }

  //Init the app (load model and textures) and OpenGL
  if (!init())
    printf("Could not Load the Model\n");

  printf ("Vendor: %s\n", glGetString (GL_VENDOR));
  printf ("Renderer: %s\n", glGetString (GL_RENDERER));
  printf ("Version: %s\n", glGetString (GL_VERSION));
  printf ("GLSL: %s\n", glGetString (GL_SHADING_LANGUAGE_VERSION));

  //return from main loop
  glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_GLUTMAINLOOP_RETURNS);
  glutMainLoop();

  }
  catch(int n) {
    if ( n == 999 )
      printf( "Finished; exiting\n" );
    else
      printf( "Unknown exception: %d", n );
  }
  catch(...) {
    printf( "Unknown exception" );
  }

  // cleaning up
  textureIdMap.clear();  

  // clear myMeshes stuff
  for (unsigned int i = 0; i < myMeshes.size(); ++i) {		
    glDeleteVertexArrays(1,&(myMeshes[i].vao));
    glDeleteTextures(1,&(myMeshes[i].texIndex));
    glDeleteBuffers(1,&(myMeshes[i].uniformBlockIndex));
  }
  // delete buffers
  glDeleteBuffers(1,&matricesUniBuffer);

  return(0);
}
